#include "hip/hip_runtime.h"
#pragma comment ( lib, "cufft.lib" )

#include <iostream>
#include <algorithm>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include ""
#include "utils.h"

#define D_MEM_CHUNKS 2
#define INNER_RADIUS 7
#define OUTER_RADIUS 3 * INNER_RADIUS
#define B1 0.238f
#define B2 0.365f
#define D1 0.267f
#define D2 0.445f
#define ALPHA_N 0.028f
#define ALPHA_M 0.147f
#define LOG_RES 8
#define DIMENSION 256
#define NUM_FIELDS 2;

using namespace std;

/* 
 * Complex type definition taken from the cuFFT samples.
 */
typedef float2 Complex;
static __device__ __host__ inline Complex ComplexAdd(Complex, Complex);
static __device__ __host__ inline Complex ComplexScale(Complex, float);
static __device__ __host__ inline Complex ComplexMul(Complex, Complex);

void fft(int, int, Complex*);
void fft2(int, int, Complex*, hipfftHandle);
Complex* besselJ(int, float&, hipfftHandle);

__global__ void draw_field(unsigned char *data, Complex *curr_field, int *color_shift, int *color_scale, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n) {
		int place = idx * 3;
		float s = curr_field[idx].x;
		for(int k=0; k<3; ++k) {
            data[place+k] = max(0, min(255, (int)(color_shift[k] + color_scale[k]*s)));
        }
	}
}

__global__ void fieldKernel(Complex* a, Complex* b, Complex* c, int n)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n) {
		float ax = a[idx].x;
		float ay = a[idx].y;
		float bx = b[idx].x;
		float by = b[idx].y;
		float t = ax * (bx + by);
		c[idx].x = t - by*(ax+ay);
		c[idx].y = t + bx*(ay-ax);
	}
}

//Precalculate multipliers for m,n
void initialize_MN(Complex* M, Complex* N, float inner_w, float outer_w){ 
    for(int i=0; i<DIMENSION; ++i) {
        for(int j=0; j<DIMENSION; ++j) {
            N[i*DIMENSION+j].x = outer_w * (N[i*DIMENSION+j].x - M[i*DIMENSION+j].x);
            N[i*DIMENSION+j].y = outer_w * (N[i*DIMENSION+j].y - M[i*DIMENSION+j].y);
            M[i*DIMENSION+j].x *= inner_w;
            M[i*DIMENSION+j].y *= inner_w;
        }
    }
}


float sigma(float x, float a, float alpha) {
    return (float)( 1.0 / (1.0 + exp(-4.0/alpha * (x - a))));
}

float sigma_2(float x, float a, float b) {
    return (float)( sigma(x, a, ALPHA_N) * (1.0f - sigma(x, b, ALPHA_N)));
}

float lerp(float a, float b, float t) {
    return (float)( (1.0f-t)*a + t*b);
}

float S(float n,float m) {
    float alive = sigma(m, 0.5f, ALPHA_M);
    return sigma_2(n, lerp(B1, D1, alive), lerp(B2, D2, alive));
}

void field_multiply(Complex* d_A, Complex* d_B, Complex* d_C, int mThreads) {
	hipError_t error;

	error = hipGetLastError();
	fieldKernel<<<(DIMENSION*DIMENSION+mThreads-1) / mThreads, mThreads>>>(d_A, d_B, d_C, DIMENSION*DIMENSION);
	error = hipGetLastError();
	if (error != hipSuccess) {
		cout << hipGetErrorString(error) << endl;
	}

	hipDeviceSynchronize();
}

//Applies the kernel to the image
void step(Complex** fields, int &current_field, Complex* M, Complex* N, Complex* M_buffer, Complex* N_buffer, int mThreads, hipfftHandle plan) {
    hipError_t error;
	Complex *d_cur_field, *d_M, *d_N, *d_M_buffer, *d_N_buffer;
    //Read in fields
    Complex* cur_field = fields[current_field];
    current_field = (current_field + 1) % 2;
    Complex* next_field = fields[current_field];
    
    //Clear imaginary values from cur_field
    for(int i=0; i<DIMENSION; ++i) {
        for(int j=0; j<DIMENSION; ++j) {
            cur_field[i*DIMENSION+j].y = 0.0f;
        }
    }
    
	error = hipMalloc((void**)&d_cur_field, DIMENSION*DIMENSION*sizeof(Complex));
	if (error != hipSuccess) {cout << hipGetErrorString(error) << endl;}
	error = hipMalloc((void**)&d_M, DIMENSION*DIMENSION*sizeof(Complex));
	if (error != hipSuccess) {cout << hipGetErrorString(error) << endl;}
	error = hipMalloc((void**)&d_N, DIMENSION*DIMENSION*sizeof(Complex));
	if (error != hipSuccess) {cout << hipGetErrorString(error) << endl;}
	error = hipMalloc((void**)&d_M_buffer, DIMENSION*DIMENSION*sizeof(Complex));
	if (error != hipSuccess) {cout << hipGetErrorString(error) << endl;}
	error = hipMalloc((void**)&d_N_buffer, DIMENSION*DIMENSION*sizeof(Complex));
	if (error != hipSuccess) {cout << hipGetErrorString(error) << endl;}

	error = hipMemcpy(d_cur_field, cur_field, DIMENSION*DIMENSION*sizeof(Complex), hipMemcpyHostToDevice);
	if (error != hipSuccess) {cout << hipGetErrorString(error) << endl;}
	error = hipMemcpy(d_M, M, DIMENSION*DIMENSION*sizeof(Complex), hipMemcpyHostToDevice);
	if (error != hipSuccess) {cout << hipGetErrorString(error) << endl;}
	error = hipMemcpy(d_N, N, DIMENSION*DIMENSION*sizeof(Complex), hipMemcpyHostToDevice);
	if (error != hipSuccess) {cout << hipGetErrorString(error) << endl;}
	error = hipMemcpy(d_M_buffer, M_buffer, DIMENSION*DIMENSION*sizeof(Complex), hipMemcpyHostToDevice);
	if (error != hipSuccess) {cout << hipGetErrorString(error) << endl;}
	error = hipMemcpy(d_N_buffer, N_buffer, DIMENSION*DIMENSION*sizeof(Complex), hipMemcpyHostToDevice);
	if (error != hipSuccess) {cout << hipGetErrorString(error) << endl;}

    //Compute m,n fields
    fft2(1, LOG_RES, d_cur_field, plan);
    field_multiply(d_cur_field, d_M, d_M_buffer, mThreads);
    fft2(-1, LOG_RES, d_M_buffer, plan);
    field_multiply(d_cur_field, d_N, d_N_buffer, mThreads);
    fft2(-1, LOG_RES, d_N_buffer, plan);
    
	error = hipMemcpy(M_buffer, d_M_buffer, DIMENSION*DIMENSION*sizeof(Complex), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {cout << hipGetErrorString(error) << endl;}
	error = hipMemcpy(N_buffer, d_N_buffer, DIMENSION*DIMENSION*sizeof(Complex), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {cout << hipGetErrorString(error) << endl;}

    //Step s
    for(int i=0; i<DIMENSION; ++i) {
        for(int j=0; j<DIMENSION; ++j) {
            next_field[i*DIMENSION+j].x = S(N_buffer[i*DIMENSION+j].x, M_buffer[i*DIMENSION+j].x);
        }
    }

	hipFree(d_cur_field);
	hipFree(d_M);
	hipFree(d_N);
	hipFree(d_M_buffer);
	hipFree(d_N_buffer);
}

//Extract image data
void draw_field(Complex** fields, int &current_field, int* color_shift, int* color_scale, int mThreads) {
    //unsigned char data [DIMENSION * DIMENSION * 3];            
    //int image_ptr = 0;
    hipError_t error;

    Complex* cur_field = fields[current_field];
    
	int n = DIMENSION*DIMENSION;

	unsigned char *h_data = new unsigned char[n * 3];
	unsigned char *d_data;
	error = hipMalloc((void**)&d_data, n*3*sizeof(unsigned char));
	if (error != hipSuccess) {
		cout << hipGetErrorString(error) << endl;
	}
	error = hipMemset(d_data, 0, n*3*sizeof(unsigned char));
	if (error != hipSuccess) {
		cout << hipGetErrorString(error) << endl;
	}

	Complex *d_field;
	error = hipMalloc((void**)&d_field, n*sizeof(Complex));
	if (error != hipSuccess) {
		cout << hipGetErrorString(error) << endl;
	}
	error = hipMemcpy(d_field, cur_field, n*sizeof(Complex), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		cout << hipGetErrorString(error) << endl;
	}

	int *d_color_scale;
	error = hipMalloc((void**)&d_color_scale, 3*sizeof(int));
	if (error != hipSuccess) {
		cout << hipGetErrorString(error) << endl;
	}
	error = hipMemcpy(d_color_scale, color_scale, 3*sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		cout << hipGetErrorString(error) << endl;
	}

	int *d_color_shift;
	error = hipMalloc((void**)&d_color_shift, 3*sizeof(int));
	if (error != hipSuccess) {
		cout << hipGetErrorString(error) << endl;
	}
	error = hipMemcpy(d_color_shift, color_shift, 3*sizeof(int), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		cout << hipGetErrorString(error) << endl;
	}

	error = hipGetLastError();
	draw_field<<<(n+mThreads-1) / mThreads, mThreads>>>(d_data, d_field, d_color_shift, d_color_scale, n);

	hipDeviceSynchronize();

	error = hipGetLastError();
	if (error != hipSuccess) {
		cout << hipGetErrorString(error) << endl;
	}

	error = hipMemcpy(h_data, d_data, n*3*sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		cout << hipGetErrorString(error) << endl;
	}
	
	/*
    for(int i=0; i<DIMENSION; ++i) {
        for(int j=0; j<DIMENSION; ++j) {
            float s = cur_field[i*DIMENSION+j];
        
            for(int k=0; k<3; ++k) {
                data[image_ptr++] = max(0, min(255, (int)(color_shift[k] + color_scale[k]*s)));
            }
        }
    }
	*/
	
    createBMP(DIMENSION, DIMENSION, h_data, DIMENSION*DIMENSION*4, "temp.bmp");
	hipFree(d_data);
	delete [] h_data;
	hipFree(d_field);
	hipFree(d_color_scale);
	hipFree(d_color_shift);
}

//Initialize field to x
void clear_field(float x, Complex** fields, int &current_field) {
    Complex* cur_field = fields[current_field];

    for(int i=0; i<DIMENSION; ++i) {
        for(int j=0; j<DIMENSION; ++j) {
          cur_field[i*DIMENSION+j].x = x;
		  cur_field[i*DIMENSION+j].y = 0.0f;
        }
    }
}


//Place a bunch of speckles on the field
void add_speckles(int count, float intensity, Complex **fields, int &current_field) {
    Complex* cur_field = fields[current_field];

	srand(time(NULL));

    for(int i=0; i<count; ++i) {
        int u = (int)(rand() % (DIMENSION-INNER_RADIUS) + 1);
        int v = (int)(rand() % (DIMENSION-INNER_RADIUS) + 1);
        for(int x=0; x<INNER_RADIUS; ++x) {
            for(int y=0; y<INNER_RADIUS; ++y) {
                cur_field[(u+x)*DIMENSION+v+y].x = intensity;
            }
        }
    }
}

int main() {
	//Coloring stuff
	int color_shift[3] = {0, 0, 0};
	int color_scale[3] = {256, 256, 256};

	//Buffers
	int field_dims[2] = {DIMENSION, DIMENSION};
	int field_size = field_dims[0] * field_dims[1];
	Complex* fields[2];

	for(int i=0; i < 2; ++i) {
		fields[i] = (Complex *)malloc(sizeof(Complex) * field_size);
	}
	int current_field = 0;
	Complex* M_buffer = (Complex *)malloc(sizeof(Complex) * field_size); // old version was a two dimensional array [256, 256]
	Complex* N_buffer = (Complex *)malloc(sizeof(Complex) * field_size);

	int d;
	hipDeviceProp_t prop;
	hipGetDevice(&d);
	hipGetDeviceProperties(&prop, d);
	int mThreads = prop.maxThreadsDim[0];
	int mBlocks  = prop.maxGridSize[0];
	int mElemnts = prop.totalGlobalMem / (D_MEM_CHUNKS * sizeof(float));

	hipfftHandle plan;
	int dims[2] = {DIMENSION, DIMENSION};
	if (hipfftPlanMany(&plan, 2, dims, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, 1) != HIPFFT_SUCCESS){
		cout << "CUFFT Error: Unable to create plan" << endl;
	}
	if (cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE)!= HIPFFT_SUCCESS){
		cout << "CUFFT Error: Unable to set compatibility mode to native" << endl;	
	}

	float inner_width;
	float outer_width;
	Complex* M = besselJ(INNER_RADIUS, inner_width, plan);
    Complex* N = besselJ(OUTER_RADIUS, outer_width, plan);
    
    float inner_w = (float)1.0f / inner_width;
    float outer_w = (float)1.0f / (outer_width - inner_width);

	initialize_MN(M, N, inner_w, outer_w);

	clear_field(0.0f, fields, current_field);
	add_speckles(300, 1.0f, fields, current_field);
	
	for (int i=0; i<100; i++) {
		step(fields, current_field, M, N, M_buffer, N_buffer, mThreads, plan);
		draw_field(fields, current_field, color_shift, color_scale, mThreads);
	}
	
	free(fields[0]);
	free(fields[1]);
	free(M_buffer);
	free(N_buffer);
	free(M);
	free(N);
	
	hipfftDestroy(plan);

	return 0;
}

//FFT
void fft(int dir, int m, Complex* a) {
	int nn,i,i1,j,k,i2,l,l1,l2;
	float c1,c2,t1,t2,u1,u2,z;
	Complex t;
    /* Calculate the number of points */
    nn = DIMENSION;
    
    /* Do the bit reversal */
    i2 = nn >> 1;
    j = 0;
    for (i=0;i<nn-1;i++) {
      if (i < j) {
         t = a[i];
         a[i] = a[j];
         a[j] = t;
      }
      k = i2;
      while (k <= j) {
         j -= k;
         k >>= 1;
      }
      j += k;
    }
    
    /* Compute the FFT */
    c1 = -1.0f;
    c2 = 0.0f;
    l2 = 1;
    for (l=0;l<m;l++) {
      l1 = l2;
      l2 <<= 1;
      u1 = 1.0f;
      u2 = 0.0f;
      for (j=0;j<l1;j++) {
         for (i=j;i<nn;i+=l2) {
            i1 = i + l1;
			t1 = u1 * a[i1].x - u2 * a[i1].y;
            t2 = u1 * a[i1].y + u2 * a[i1].x;
            a[i1].x = a[i].x - t1;
            a[i1].y = a[i].y - t2;
            a[i].x += t1;
            a[i].y += t2;
         }
         z =  u1 * c1 - u2 * c2;
         u2 = u1 * c2 + u2 * c1;
         u1 = z;
      }
      c2 = sqrt((1.0f - c1) / 2.0f);
      if (dir == 1)
         c2 = -c2;
      c1 = sqrt((1.0f + c1) / 2.0f);
    }
    
    /* Scaling for forward transform */
    if (dir == -1) {
      float scale_f = 1.0f / nn;        
      for (i=0;i<nn;i++) {
         a[i] = ComplexScale(a[i], scale_f);
      }
    }
}

//In place 2D fft
void fft2(int dir, int m, Complex* a, hipfftHandle plan) {
	if (hipfftExecC2C(plan, (hipfftComplex*)a, (hipfftComplex*)a, (dir==1 ? HIPFFT_FORWARD : HIPFFT_BACKWARD)) != HIPFFT_SUCCESS){
		cout << "CUFFT Error: Unable to execute plan" << endl;	
	}
}

//In place 2D fft
void hostfft2(int dir, int m, Complex* a) {
  for(int i=0; i<DIMENSION; ++i) {
    fft(dir, m, &a[i*DIMENSION]);
  }
  for(int i=0; i<DIMENSION; ++i) {
    for(int j=0; j<i; ++j) {
      Complex t = a[i*DIMENSION+j];
      a[i*DIMENSION+j] = a[j*DIMENSION+i];
      a[j*DIMENSION+i] = t;
    }
  }

  for(int i=0; i<DIMENSION; ++i) {
    fft(dir, m, &a[i*DIMENSION]);
  }
}

Complex* besselJ(int radius, float &w, hipfftHandle plan) {
	int field_size = DIMENSION * DIMENSION;

    //Do this in a somewhat stupid way
    Complex* a = (Complex *)malloc(sizeof(Complex) * field_size);;
    w = 0.0f;
    for(int i=0; i<DIMENSION; ++i) {
        for(int j=0; j<DIMENSION; ++j) {
            float ii = (float)((i + DIMENSION/2) % DIMENSION) - DIMENSION/2;
            float jj = (float)((j + DIMENSION/2) % DIMENSION) - DIMENSION/2;
            
            float r = sqrt(ii*ii + jj*jj) - radius;
            float v = 1.0f / (1.0f + exp(LOG_RES * r));
            
            w += v;
            a[i*DIMENSION+j].x = v;
			a[i*DIMENSION+j].y = 0.0f;
        }
    }
    
    hostfft2(1, LOG_RES, a);

	return a;
}

////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex scale
static __device__ __host__ inline Complex ComplexScale(Complex a, float s)
{
    Complex c;
    c.x = s * a.x;
    c.y = s * a.y;
    return c;
}

// Complex pointwise multiplication
/*
static __global__ void ComplexPointwiseMulAndScale(Complex *a, const Complex *b, int size, float scale)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < size; i += numThreads)
    {
        a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);
    }
}
*/
